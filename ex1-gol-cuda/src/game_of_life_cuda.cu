#include "hip/hip_runtime.h"
#include "common.h"

// CUDA error checking - derived from [https://stackoverflow.com/a/14038590]
#define cuda_error_check(x) {cuda_examine(x, __FILE__, __LINE__);}
inline void cuda_examine(hipError_t code, const char * file, int line, \
                         bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "cuda_error_check: (%s:%d) %s\n", file, line, \
            hipGetErrorString(code));

    if (abort)
    {
      exit(code);
    }
  }
}

// debug flags
// - `debug_verbose != 0` will annotate, to stderr, the program as it is
//   executed
// - `debug_timing != 0` will annotate, to stderr, the timing variables as
//   they are calculated
// - `debug_visual != 0` will annotate, to stderr, the ascii visualisation of
//   grid variables as they are intialised and updated
const int debug_verbose = 1;
const int debug_timing = 1;
const int debug_visual = 1;

// verbose macro
#define verbose(format, ...)                                \
  if (debug_verbose) {                                      \
    fprintf(stderr, "[verbose] "format"\n", ##__VA_ARGS__); \
  }

// timing macro
#define timing(format, ...)                                 \
  if (debug_timing) {                                       \
    fprintf(stderr, "[timing] "format"\n", ##__VA_ARGS__);  \
  }

// visual macro
#define visual(current_step, grid, n, m, format, ...)       \
  if (debug_visual) {                                       \
    fprintf(stderr, "[visual] "format"\n", ##__VA_ARGS__);  \
    visualise_ascii(current_step, grid, n, m);              \
  }

__global__ void gpu_game_of_life_step(int *current_grid, int *next_grid, \
                                      int n, int m)
{
  // indexing variables
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int i = idx / m;
  int j = idx % m;

  // only perform kernel for valid cell indexes
  if ((i < n) && (j < m))
  {
    // neighbourhood variables
    int neighbours;
    int n_i[8], n_j[8];

    // count the number of neighbours, clockwise around the current cell.
    neighbours = 0;
    n_i[0] = i - 1; n_j[0] = j - 1;
    n_i[1] = i - 1; n_j[1] = j;
    n_i[2] = i - 1; n_j[2] = j + 1;
    n_i[3] = i;     n_j[3] = j + 1;
    n_i[4] = i + 1; n_j[4] = j + 1;
    n_i[5] = i + 1; n_j[5] = j;
    n_i[6] = i + 1; n_j[6] = j - 1;
    n_i[7] = i;     n_j[7] = j - 1;

    if (n_i[0] >= 0 && n_j[0] >= 0                                    \
        && current_grid[n_i[0] * m + n_j[0]] == ALIVE) neighbours++;
    if (n_i[1] >= 0                                                   \
        && current_grid[n_i[1] * m + n_j[1]] == ALIVE) neighbours++;
    if (n_i[2] >= 0 && n_j[2] < m                                     \
        && current_grid[n_i[2] * m + n_j[2]] == ALIVE) neighbours++;
    if (n_j[3] < m                                                    \
        && current_grid[n_i[3] * m + n_j[3]] == ALIVE) neighbours++;
    if (n_i[4] < n && n_j[4] < m                                      \
        && current_grid[n_i[4] * m + n_j[4]] == ALIVE) neighbours++;
    if (n_i[5] < n                                                    \
        && current_grid[n_i[5] * m + n_j[5]] == ALIVE) neighbours++;
    if (n_i[6] < n && n_j[6] >= 0                                     \
        && current_grid[n_i[6] * m + n_j[6]] == ALIVE) neighbours++;
    if (n_j[7] >= 0                                                   \
        && current_grid[n_i[7] * m + n_j[7]] == ALIVE) neighbours++;

    if (current_grid[i*m + j] == ALIVE && (neighbours == 2 || neighbours == 3))
    {
      next_grid[i*m + j] = ALIVE;
    }
    else if (current_grid[i*m + j] == DEAD && neighbours == 3)
    {
      next_grid[i*m + j] = ALIVE;
    }
    else
    {
      next_grid[i*m + j] = DEAD;
    }
  }
}

/*
  Implements the game of life on a grid of size `n` times `m`, starting from
  the `initial_state` configuration.

  If `nsteps` is positive, returns the last state reached.
*/
int* gpu_game_of_life(const int *initial_state, int n, int m, int nsteps, \
                      float *kernel_time)
{
  // cuda kernel parameters - uses least amount of blocks required
  const int n_threads = 1024;
  const int n_blocks = ((n * m - 1) / n_threads) + 1;

  verbose ("CUDA: <n_blocks> = %i, <n_threads> = %i", n_blocks, n_threads);

  // allocate gpu memory
  int *grid;
  int *updated_grid;

  cuda_error_check(hipMalloc(&grid, sizeof(int) * n * m));
  cuda_error_check(hipMalloc(&updated_grid, sizeof(int) * n * m));

  verbose ("CUDA: <grid>, <updated_grid> memory allocated (GPU)");

  // copy initial state to gpu memory
  cuda_error_check(hipMemcpy(grid, initial_state, sizeof(int) * n * m, \
                              hipMemcpyHostToDevice));

  verbose ("CUDA: copied <intial_state> (CPU) to <grid> (GPU)");

  // prepare kernel timing variables
  *kernel_time = 0.0;

  hipEvent_t kernel_start, kernel_stop;
  cuda_error_check(hipEventCreate(&kernel_start));
  cuda_error_check(hipEventCreate(&kernel_stop));
  float kernel_time_step = 0.0;

  verbose ("CUDA: <kernel_start>, <kernel_stop> CUDA events defined");

  // initialise game_of_life loop
  int current_step = 0;

  while (current_step != nsteps)
  {
    current_step++;

    verbose("CUDA: <%i> GOL step started", current_step);

    // initialise timing of kernel execution
    cuda_error_check(hipEventRecord(kernel_start));

    verbose("CUDA: <%i> timing intialised", current_step);

    // calculate next state of GOL using CUDA kernel across grid
    gpu_game_of_life_step<<<n_blocks, n_threads>>>(grid, updated_grid, n, m);

    verbose("CUDA: <%i> next GOL state calculated", current_step);

    // finalise timing of kernel execution
    cuda_error_check(hipEventRecord(kernel_stop));
    cuda_error_check(hipDeviceSynchronize(kernel_stop));

    // swap current and updated grid
    {
      int *tmp = grid;
      grid = updated_grid;
      updated_grid = tmp;
    }

    verbose("CUDA: <%i> grids swapped", current_step);

    // calculate timing of kernel execution
    cuda_error_check(hipEventElapsedTime(&kernel_time_step, kernel_start, \
                                          kernel_stop));
    *kernel_time += kernel_time_step;

    timing("CUDA: <step_time, %i> = %f [ms]", current_step, kernel_time_step);

    // debug: visualise `grid` after current step
    if (debug_visual)
    {
      visual(current_step, grid, n, m, "<grid, %i> = ", current_step);
    }

    verbose("CUDA: <%i> GOL step finished", current_step);
  }

  verbose("CUDA: GOL loop finished");

  // copy final state to cpu memory
  int *final_state = (int *) malloc(sizeof(int) * n * m);

  if (final_state == NULL)
  {
    fprintf(stderr, "error while allocating memory for <final_state>\n");
    exit(1);
  }

  cuda_error_check(hipMemcpy(final_state, grid, sizeof(int) * n * m, \
                              hipMemcpyDeviceToHost));

  verbose ("CUDA: copied <grid> (GPU) to <final_state> (CPU)");

  // free gpu memory
  hipFree(updated_grid);
  hipFree(grid);

  verbose("CUDA: <grid>, <updated_grid> memory freed (GPU)");

  return final_state;
}

// write timing data for gpu CUDA code to file
int gpu_write_timing(struct Options const * opt, float const elapsed_time, \
                     float const kernel_time)
{
  FILE *file = NULL;
  char filename[200];
  int ierr = 0;

  // create filename for given options
  sprintf(filename, "output/timing-gpu-cuda.n-%i.m-%i.nsteps-%i.txt",  \
          opt->n, opt->m, opt->nsteps);

  printf("writing gpu timing data to filename: %s\n", filename);

  // open file
  file = fopen(filename, "w");

  if (file == NULL)
  {
    fprintf(stderr, "cannot open filename: %s\n", filename);
    ierr = 1;
  }
  else
  {
    // write timing data
    fprintf(file, "# gpu_elapsed_time, gpu_kernel_time\n");
    fprintf(file, "# [ms], [ms]\n");
    fprintf(file, "%f, %f\n", elapsed_time, kernel_time);

    // close file
    fclose(file);
  }

  return ierr;
}

// do not define the main function if this file is included somewhere else.
#ifndef INCLUDE_GPU_VERSION
int main(int argc, char **argv)
{
  // debug: verbose
  verbose("<debug_verbose> = on");
  if (debug_timing) verbose("<debug_timing> = on");
  if (debug_visual) verbose("<debug_visual> = on");

  // define timing variables
  struct timeval start;
  struct timeval gol_start;

  // initialise timing of entire program execution
  start = init_time();

  verbose("program timing initialised");

  // read input
  struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
  getinput(argc, argv, opt);

  verbose("read input");

  // define parameter variables
  const int n = opt->n;
  const int m = opt->m;
  const int nsteps = opt->nsteps;

  verbose("parameters defined: <n> = %i, <m> = %i, <nsteps> = %i", \
          n, m, nsteps);

  // allocate memory for `initial_state` variable
  int *initial_state = (int *) malloc(sizeof(int) * n * m);

  if (initial_state == NULL)
  {
    fprintf(stderr, "error while allocating memory for <initial_state>\n");
    return -1;
  }

  verbose("<initial_state> memory allocated: sizeof(int) * %i", n * m);

  // generate initial conditions
  generate_IC(opt->iictype, initial_state, n, m);

  verbose("<initial_state> initial conditions generated");

  // debug: visualise `intial_state` after initial conditions
  visual(0, initial_state, n, m, "<initial_state> = ");

  // initialise timing of GOL simulation
  gol_start = init_time();

  verbose("GOL simulation timing initialised");

  // calculate `final_state` (and record kernel time)
  float kernel_time = 0.0;
  int *final_state = gpu_game_of_life(initial_state, n, m, nsteps,
                                      &kernel_time);

  // calculate time for GOL simulation
  float elapsed_time = get_elapsed_time(gol_start);
  timing("<elapsed_time> = %f [ms]", elapsed_time);

  // calculate kernel time
  timing("<kernel_time> = %f [ms] / (%f%%)", kernel_time,
         100.0*kernel_time/elapsed_time);

  verbose("GOL simulation timing finished");

  // write timing to file
  gpu_write_timing(opt, elapsed_time, kernel_time);

  verbose("<elapsed_time> written to file");

  // debug: visualise `final_state` after loop completion
  visual(nsteps, final_state, n, m, "<final_state> = ");

  // free cpu memory
  free(final_state);
  free(initial_state);
  free(opt);

  verbose("memory freed");

  // debug: calculate time for entire program execution
  float total_time = get_elapsed_time(start);
  timing("<total_time> = %f [ms]", total_time);

  verbose("program timing finished");

  return 0;
}
#endif
