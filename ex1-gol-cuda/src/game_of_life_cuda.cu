#include "hip/hip_runtime.h"
#include "common.h"

// CUDA error checking - derived from [https://stackoverflow.com/a/14038590]
#define cuda_error_check(x) {cuda_examine(x, __FILE__, __LINE__);}
inline void cuda_examine(hipError_t code, const char * file, int line, \
                         bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "cuda_error_check: (%s:%d) %s\n", file, line, \
            hipGetErrorString(code));

    if (abort)
    {
      exit(code);
    }
  }
}

// debug flags
// - `debug_verbose != 0` will annotate, to stderr, the program as it is
//   executed
// - `debug_timing != 0` will annotate, to stderr, the timing variables as
//   they are calculated
// - `debug_visual != 0` will annotate, to stderr, the ascii visualisation of
//   grid variables as they are intialised and updated
const int debug_verbose = 1;
const int debug_timing = 1;
const int debug_visual = 1;

// verbose macro
#define verbose(format, ...)                                \
  if (debug_verbose) {                                      \
    fprintf(stderr, "[verbose] "format"\n", ##__VA_ARGS__); \
  }

// timing macro
#define timing(format, ...)                                 \
  if (debug_timing) {                                       \
    fprintf(stderr, "[timing] "format"\n", ##__VA_ARGS__);  \
  }

// visual macro
#define visual(current_step, grid, n, m, format, ...)       \
  if (debug_visual) {                                       \
    fprintf(stderr, "[visual] "format"\n", ##__VA_ARGS__);  \
    visualise_ascii(current_step, grid, n, m);              \
  }

__global__ void gpu_game_of_life_step(int *current_grid, int *next_grid, \
                                      int n, int m)
{
  // indexing variables
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int i = idx / m;
  int j = idx % m;

  // only perform kernel for valid cell indexes
  if ((i < n) && (j < m))
  {
    // neighbourhood variables
    int neighbours;
    int n_i[8], n_j[8];

    // count the number of neighbours, clockwise around the current cell.
    neighbours = 0;
    n_i[0] = i - 1; n_j[0] = j - 1;
    n_i[1] = i - 1; n_j[1] = j;
    n_i[2] = i - 1; n_j[2] = j + 1;
    n_i[3] = i;     n_j[3] = j + 1;
    n_i[4] = i + 1; n_j[4] = j + 1;
    n_i[5] = i + 1; n_j[5] = j;
    n_i[6] = i + 1; n_j[6] = j - 1;
    n_i[7] = i;     n_j[7] = j - 1;

    if (n_i[0] >= 0 && n_j[0] >= 0                                    \
        && current_grid[n_i[0] * m + n_j[0]] == ALIVE) neighbours++;
    if (n_i[1] >= 0                                                   \
        && current_grid[n_i[1] * m + n_j[1]] == ALIVE) neighbours++;
    if (n_i[2] >= 0 && n_j[2] < m                                     \
        && current_grid[n_i[2] * m + n_j[2]] == ALIVE) neighbours++;
    if (n_j[3] < m                                                    \
        && current_grid[n_i[3] * m + n_j[3]] == ALIVE) neighbours++;
    if (n_i[4] < n && n_j[4] < m                                      \
        && current_grid[n_i[4] * m + n_j[4]] == ALIVE) neighbours++;
    if (n_i[5] < n                                                    \
        && current_grid[n_i[5] * m + n_j[5]] == ALIVE) neighbours++;
    if (n_i[6] < n && n_j[6] >= 0                                     \
        && current_grid[n_i[6] * m + n_j[6]] == ALIVE) neighbours++;
    if (n_j[7] >= 0                                                   \
        && current_grid[n_i[7] * m + n_j[7]] == ALIVE) neighbours++;

    if (current_grid[i*m + j] == ALIVE && (neighbours == 2 || neighbours == 3))
    {
      next_grid[i*m + j] = ALIVE;
    }
    else if (current_grid[i*m + j] == DEAD && neighbours == 3)
    {
      next_grid[i*m + j] = ALIVE;
    }
    else
    {
      next_grid[i*m + j] = DEAD;
    }
  }
}

/*
  Implements the game of life on a grid of size `n` times `m`, starting from
  the `initial_state` configuration.

  If `nsteps` is positive, returns the last state reached.
*/
int* gpu_game_of_life(const int *initial_state, int n, int m, int nsteps, \
                      float *kernel_time)
{
  // cuda kernel parameters - uses least amount of blocks required
  const int n_threads = 1024;
  const int n_blocks = ((n * m - 1) / n_threads) + 1;

  // allocate gpu memory
  int *grid;
  int *updated_grid;

  cuda_error_check(hipMalloc(&grid, sizeof(int) * n * m));
  cuda_error_check(hipMalloc(&updated_grid, sizeof(int) * n * m));

  // copy initial state to gpu memory
  cuda_error_check(hipMemcpy(grid, initial_state, sizeof(int) * n * m, \
                              hipMemcpyHostToDevice));

  // prepare kernel timing variables
  *kernel_time = 0.0;

  hipEvent_t kernel_start, kernel_stop;
  cuda_error_check(hipEventCreate(&kernel_start));
  cuda_error_check(hipEventCreate(&kernel_stop));
  float kernel_time_step = 0.0;

  // initialise game_of_life loop
  int current_step = 0;
  int *tmp = NULL;

  while (current_step != nsteps)
  {
    current_step++;

    // Uncomment the following line if you want to print the state at every step
    // visualise(opt->ivisualisetype, current_step, grid, n, m);

    // execute game_of_life_step cuda kernel
    cuda_error_check(hipEventRecord(kernel_start));
    gpu_game_of_life_step<<<n_blocks, n_threads>>>(grid, updated_grid, n, m);
    cuda_error_check(hipEventRecord(kernel_stop));
    cuda_error_check(hipDeviceSynchronize());
    cuda_error_check(hipEventElapsedTime(&kernel_time_step, kernel_start, \
                                          kernel_stop));
    *kernel_time += kernel_time_step;

    // swap current and updated grid
    tmp = grid;
    grid = updated_grid;
    updated_grid = tmp;
  }

  // copy final state to cpu memory
  int *final_state = (int *) malloc(sizeof(int) * n * m);

  if (!final_state)
  {
    printf("gpu_game_of_life: error while allocating memory.\n");
    exit(1);
  }

  cuda_error_check(hipMemcpy(final_state, grid, sizeof(int) * n * m, \
                              hipMemcpyDeviceToHost));

  // free gpu memory
  hipFree(updated_grid);
  hipFree(grid);
  hipFree(tmp);

  return final_state;
}

// write timing data for gpu CUDA code to file
int gpu_write_timing(struct Options const * opt, float const elapsed_time, \
                     float const kernel_time)
{
  FILE *file = NULL;
  char filename[200];
  int ierr = 0;

  // create filename for given options
  sprintf(filename, "output/timing-gpu-cuda.n-%i.m-%i.nsteps-%i.txt",  \
          opt->n, opt->m, opt->nsteps);

  printf("writing gpu timing data to filename: %s\n", filename);

  // open file
  file = fopen(filename, "w");

  if (file == NULL)
  {
    fprintf(stderr, "cannot open filename: %s\n", filename);
    ierr = 1;
  }
  else
  {
    // write timing data
    fprintf(file, "# gpu_elapsed_time, gpu_kernel_time\n");
    fprintf(file, "# [ms], [ms]\n");
    fprintf(file, "%f, %f\n", elapsed_time, kernel_time);

    // close file
    fclose(file);
  }

  return ierr;
}

// do not define the main function if this file is included somewhere else.
#ifndef INCLUDE_GPU_VERSION
int main(int argc, char **argv)
{
  // debug: verbose
  verbose("<debug_verbose> = on");
  if (debug_timing) verbose("<debug_timing> = on");
  if (debug_visual) verbose("<debug_visual> = on");

  // define timing variables
  struct timeval start;
  struct timeval gol_start;

  // initialise timing of entire program execution
  start = init_time();

  verbose("program timing initialised");

  // read input
  struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
  getinput(argc, argv, opt);

  verbose("read input");

  // define parameter variables
  const int n = opt->n;
  const int m = opt->m;
  const int nsteps = opt->nsteps;

  verbose("parameters defined: <n> = %i, <m> = %i, <nsteps> = %i", \
          n, m, nsteps);

  // allocate memory for `initial_state` variable
  int *initial_state = (int *) malloc(sizeof(int) * n * m);
  //int *final_state = (int *) malloc(sizeof(int) * n * m);

  if (initial_state == NULL)
  {
    fprintf(stderr, "error while allocating memory for <initial_state>\n");
    return -1;
  }

  verbose("<initial_state> memory allocated: sizeof(int) * %i", n * m);

  // generate initial conditions
  generate_IC(opt->iictype, initial_state, n, m);

  verbose("<initial_state> initial conditions generated");

  // debug: visualise `intial_state` after initial conditions
  visual(0, initial_state, n, m, "<initial_state> = ");

  // initialise timing of GOL simulation
  gol_start = init_time();

  verbose("GOL simulation timing initialised");

  // calculate `final_state` (and record kernel time)
  float kernel_time = 0.0;
  final_state = gpu_game_of_life(initial_state, n, m, nsteps, &kernel_time);


  // calculate time for GOL simulation
  float elapsed_time = get_elapsed_time(gol_start);
  timing("<elapsed_time> = %f [ms]", elapsed_time);

  // calculate kernel time
  timing("<kernel_time> = %f [ms] / (%f%%)", kernel_time,
         100.0*kernel_time/elapsed_time);

  verbose("GOL simulation timing finished");

  // write timing to file
  gpu_write_timing(opt, elapsed_time, kernel_time);

  verbose("<elapsed_time> written to file");

  // debug: visualise `final_state` after loop completion
  visual(nsteps, final_state, n, m, "<final_state> = ");

  // free cpu memory
  free(final_state);
  free(initial_state);
  free(opt);

  verbose("memory freed");

  // debug: calculate time for entire program execution
  float total_time = get_elapsed_time(start);
  timing("<total_time> = %f [ms]", total_time);

  verbose("program timing finished");

  return 0;
}
#endif
